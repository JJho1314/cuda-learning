

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> // cpu的malloc函数

void cpu(int *a , int N){
    for(int i=0;i<N;i++){
        a[i] = i;
    }
    printf("hello cpu\n");
}
// global 将在gpu上运行并可全局调用
__global__ void gpu(int *a, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<N){
        a[i]*=2; // 放大2倍
    }
}

// 验证
bool check(int *a, int N){
    for(int i = 0;i<N;i++){
        if(a[i] != 2*i) return false;
    }
    return true;
}


int main(){
    const int N = 2 << 5; //二进制左移运算符。
    size_t size = N*sizeof(int);
    int *a; //取指针的地址&a
    hipMallocManaged(&a, size); // 既可以被cpu使用也可以被gpu使用
    cpu(a, N);
    
    // gpu
    size_t threads = 256;
    size_t blocks = (N + threads  -1)/threads; // 算法竞赛向上取整  ceil也可
    gpu<<<blocks, threads>>>(a, N); // 每一个数都拥有一个线程
    hipDeviceSynchronize();

    check(a, N)?printf("Ok") : printf("Sorry");
    hipFree(a);
}