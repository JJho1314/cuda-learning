/*
 * @Description: 
 * @Author: HCQ
 * @Company(School): UCAS
 * @Email: 1756260160@qq.com
 * @Date: 2022-02-02 20:13:18
 * @LastEditTime: 2022-02-03 10:19:26
 * @FilePath: /cuda-learning/00hellocuda/hello-gpu.cu
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

void cpu(){
    printf("hello cpu\n");
}
// global 将在gpu上运行并可全局调用
__global__ void gpu(){
    // 通过一个表达式区分不同线程
    // blockIdx.x * blockDim.x + threadIdx.x; 
    // 只希望第一个block的第一个线程去打印
    if(blockIdx.x==0&&threadIdx.x==0){
        printf("hello gpu\n");
    }
}


int main(){
    cpu();
    gpu<<<2,3>>>();  // gpu配置 《block 线程数》
    hipDeviceSynchronize();
}